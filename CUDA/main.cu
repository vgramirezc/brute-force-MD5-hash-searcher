#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <fstream>
#include <math.h>
#include <string.h>
#include <vector>
#include "md5.h"
#include "omp.h"

using namespace std;

const int HASH_LENGTH = 32;
const int HASH_ALPHA_SIZE = 16;
const int MAX_HASHES_SIZE = 100005;
const int SIZE_MATCHES = 5e5;
int trie_size = 0;
int hashes_trie[HASH_ALPHA_SIZE*HASH_LENGTH*MAX_HASHES_SIZE];
int hash_id[HASH_LENGTH*MAX_HASHES_SIZE];
long long matches_ids[SIZE_MATCHES];

int ch_to_int( string str ){
    int len = str.size();
    int ans = 0;
    for( int i = 0; i < len; ++i ){
        if( str[ i ] < '0' || str[ i ] > '9' ) return -1;
        ans *= 10;
        ans += str[ i ] - '0';
    }
    return ans;
}

bool check_format( char* format, int format_size ){
    for( int i = 0; i < format_size; ++i ){
        if( format[ i ] != 'a' && format[ i ] != 'A' && format[ i ] != '0' )
            return false;
    }
    return true;
}

__host__ __device__ int char_to_child_id( char ch ){
    if(ch >= '0' && ch <= '9') return ch - '0';
    return ch - 'a' + 10;
}

void trie_insert( const string& hash ){
    int cur = 0;
    for(int i = 0; i < hash.size(); ++i){
        int ch = char_to_child_id( hash[i] );
        if(hashes_trie[cur*HASH_ALPHA_SIZE+ch] == 0)
            hashes_trie[cur*HASH_ALPHA_SIZE+ch] = ++trie_size;
        cur = hashes_trie[cur*HASH_ALPHA_SIZE+ch];
    }
}

void build_hashes_trie( ifstream& in_hash ){
    string h;
    while( in_hash >> h ){
        trie_insert( h );
    }
}

__host__ __device__ bool is_in_hash_trie( char * hash, int* trie, long long string_val ){
    int cur = 0;
    for(int i = 0; i < HASH_LENGTH; ++i){
        int ch = char_to_child_id( hash[i] );
        if(trie[cur*HASH_ALPHA_SIZE+ch] == 0){
            return false;
        }
        cur = trie[cur*HASH_ALPHA_SIZE+ch];
    }
    return true;
}

int get_total_strings( char* format, int format_size ){
    long long total = 1LL;
    for(int i = 0; i < format_size; ++i){
        if(format[i] == '0') total *= 10LL;
        else total *= 26LL;
    }
    return total;
}

__host__ __device__ void value_to_password( long long val, char* format, int format_size, char* pass ){
    for(int i = format_size-1; i >= 0; --i){
        int base = (format[i] == '0') ? 10 : 26;
        pass[i] = format[i] + val % base;
        val /= base;
    }
    pass[format_size] = '\0';
}

__device__ bool check_password(long long string_val, MD5 * d_encrypter, char* d_passwords, char* d_hashes, char* format, int format_size, int* d_cnt, int* d_trie, int index){
    value_to_password(string_val, format, format_size, d_passwords+((format_size+1)*index));
    (d_encrypter+index)->change_text(d_passwords+((format_size+1)*index));
    (d_encrypter+index)->hexdigest(d_hashes+((HASH_LENGTH+1)*index));
    if( is_in_hash_trie( d_hashes+((HASH_LENGTH+1)*index), d_trie, string_val ) ){
        d_cnt[index] = d_cnt[index] + 1;
        return true;
        //printf( "Hash %s matched with string %s\n", d_hashes+((HASH_LENGTH+1)*index), d_passwords+((format_size+1)*index));
    }
    return false;
}

__global__ void brute_force( int * d_cnt, int * d_trie, char * d_format, MD5* d_encrypter, char* d_passwords, char* d_hashes, long long * d_matches, int format_size, int total_threads, long long total_strings ){
    long long passwords_per_thread = (total_strings + total_threads - 1) / total_threads;
    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    long long st = passwords_per_thread * index;
    long long en = min(st + passwords_per_thread, total_strings);
    int sz_per_wi = (SIZE_MATCHES + total_threads - 1) / total_threads;
    int cur_match_pos = index * sz_per_wi;
    int max_match_pos = cur_match_pos + sz_per_wi < SIZE_MATCHES ? cur_match_pos + sz_per_wi : SIZE_MATCHES;
    for(long long i = st; i < en; ++i){
        if( check_password(i, d_encrypter, d_passwords, d_hashes, d_format, format_size, d_cnt, d_trie, index) ){
            if(cur_match_pos < max_match_pos){
                d_matches[cur_match_pos] = i;
            }
            ++cur_match_pos;
        }
    }
}

int main( int argc, char *argv[] ){
    if( argc != 5 ){
        printf( "You must pass exactly four arguments:\n" );
        printf( "  1. The password pattern\n" );
        printf( "  2. The name of the file containing the hashes.\n" );
        printf( "  3. The total number of threads.\n" );
        printf( "  4. The number of blocks.\n" );
        exit( 0 );
    }
    char * format = argv[1];
    int format_size = strlen( format );
    if( !check_format( format, format_size ) ){
        printf( "The password pattern must have the following format:\n" );
        printf( "  * a: lowercase letter [a-z].\n" );
        printf( "  * A: uppercase letter. [A-Z]\n" );
        printf( "  * 0: digit. [0-9]\n" );
        printf( "E.g.: Aaaaa00 means passwords of length 7 starting with an uppercase letter, followed by 4 lowercase letters and ending with two digits.\n" );
        exit( 0 );
    }
    ifstream in_hash( argv[2] );
    if( !in_hash ){
        printf( "Error reading file %s\n", argv[2] );
        exit( 0 );
    }
    int num_threads = ch_to_int( argv[ 3 ] );
    if( num_threads <= 0 ){
        printf( "Invalid argument [%s]. The number of threads must be a positive integer.\n", argv[3] );
        exit( 0 );
    }
    int number_of_blocks = ch_to_int( argv[ 4 ] );
    if( number_of_blocks <= 0 ){
        printf( "Invalid argument [%s]. The number of blocks must be a positive integer.\n", argv[4] );
        exit( 0 );
    }
    build_hashes_trie( in_hash );
    long long total_strings = get_total_strings( format, format_size );

    int size_cnt = sizeof(int) * num_threads;
    //Declaring pointers
    int* h_cnt;
    int* d_cnt;
    char* d_format;
    int* d_trie;
    MD5* d_encrypter;
    char* d_passwords;
    char* d_hashes;
    long long * d_matches;

    //Alloc memory
    h_cnt = (int*)malloc(size_cnt);
    hipError_t err = hipSuccess;

    err = hipMalloc( (void **) &d_cnt, size_cnt );
    if (err != hipSuccess){
        fprintf(stderr, "Error 1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_format, sizeof(format) );
    if (err != hipSuccess){
        fprintf(stderr, "Error 2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_trie, sizeof(int)*HASH_ALPHA_SIZE*HASH_LENGTH*MAX_HASHES_SIZE );
    if (err != hipSuccess){
        fprintf(stderr, "Error 3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_encrypter, sizeof(MD5)*num_threads );
    if (err != hipSuccess){
        fprintf(stderr, "Error 3.2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_passwords, sizeof(char)*(format_size+1)*num_threads );
    if (err != hipSuccess){
        fprintf(stderr, "Error 3.3 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_hashes, sizeof(char)*(HASH_LENGTH+1)*num_threads );
    if (err != hipSuccess){
        fprintf(stderr, "Error 3.4 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMalloc( (void **) &d_matches, sizeof(matches_ids) );
    if (err != hipSuccess){
        fprintf(stderr, "Error 3.5 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Initialize
    for(int i = 0; i < num_threads; ++i)
        h_cnt[i] = 0;

    //Copy host to device
    err = hipMemcpy( d_cnt, h_cnt, size_cnt, hipMemcpyHostToDevice );
    if (err != hipSuccess){
        fprintf(stderr, "Error 4 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy( d_format, format, sizeof(format), hipMemcpyHostToDevice );
    if (err != hipSuccess){
        fprintf(stderr, "Error 5 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy( d_trie, &hashes_trie, sizeof(hashes_trie), hipMemcpyHostToDevice );
    if (err != hipSuccess){
        fprintf(stderr, "Error 6 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Launch kernel
    int total_threads;
    if(num_threads <= number_of_blocks){
        total_threads = num_threads;
        printf("CUDA kernel launch with %d block(s) of %d thread(s) Total: %i\n", num_threads, 1, num_threads);
        brute_force<<<num_threads, 1>>>(d_cnt, d_trie, d_format, d_encrypter, d_passwords, d_hashes, d_matches, format_size, num_threads, total_strings);
    }
    else{
        int threads_per_block = num_threads/number_of_blocks;
        total_threads = number_of_blocks * threads_per_block;
        printf("CUDA kernel launch with %d block(s) of %d thread(s) Total: %i\n", number_of_blocks, threads_per_block, total_threads);
        brute_force<<<number_of_blocks, threads_per_block>>>(d_cnt, d_trie, d_format, d_encrypter, d_passwords, d_hashes, d_matches, format_size, total_threads, total_strings);
    }

    err = hipGetLastError();
    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Copy device to host
    err = hipMemcpy( h_cnt, d_cnt, size_cnt, hipMemcpyDeviceToHost );
    if (err != hipSuccess){
        fprintf(stderr, "Error 7 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy( matches_ids, d_matches, sizeof(matches_ids), hipMemcpyDeviceToHost );
    if (err != hipSuccess){
        fprintf(stderr, "Error 7.2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    //Free memory
    err = hipFree( d_cnt );
    if (err != hipSuccess){
        fprintf(stderr, "Error 8 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_trie );
    if (err != hipSuccess){
        fprintf(stderr, "Error 9 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_format );
    if (err != hipSuccess){
        fprintf(stderr, "Error 10 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_encrypter );
    if (err != hipSuccess){
        fprintf(stderr, "Error 11 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_passwords );
    if (err != hipSuccess){
        fprintf(stderr, "Error 12 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_hashes );
    if (err != hipSuccess){
        fprintf(stderr, "Error 13 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree( d_matches );
    if (err != hipSuccess){
        fprintf(stderr, "Error 14 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    int total_matches = 0;
    MD5 md5;
    char match_pass[format_size+1], match_hash[HASH_LENGTH];
    int sz_per_wi = (SIZE_MATCHES + total_threads - 1) / total_threads;
    for(int i = 0, idx = 0; i < total_threads; ++i, idx += sz_per_wi){
        for(int j = 0; j < h_cnt[i]; ++j){
            if(j == sz_per_wi || idx + j == SIZE_MATCHES) break;
            value_to_password( matches_ids[idx + j], format, format_size, match_pass );
            md5.change_text(match_pass);
            md5.hexdigest(match_hash);
            printf("Hash %s matched with string %s\n", match_hash, match_pass);
        }
        total_matches += h_cnt[i];
    }
    printf("Total matches %d\n", total_matches);
    return 0;
}